#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime_api.h"
#include <hiprand/hiprand_kernel.h>
#include <math_functions.h>
#include <hittable_list.h>
#include <ray.h>
#include <sphere.h>
#include <hittable_list.h>
#include <material.h>
#include <vec3.h>
#include <camera.h>
hittable** world;
hittable** objects;
camera** cam;
int object_counts = 2;

hiprandState* random_state;
// convert floating point rgb color to 8-bit integer
__device__ float clamp(double x, double a, double b) { return max(a, min(b, x)); }
__device__ int rgbToInt(double r, double g, double b) {
	r = clamp(r, 0.0f, 255.0f);
	g = clamp(g, 0.0f, 255.0f);
	b = clamp(b, 0.0f, 255.0f);
	return (int(b) << 16) | (int(g) << 8) | int(r);
}
__device__ int vectorgb(vec3 color) {
	return rgbToInt(color.x()*255,color.y()*255,color.z()*255);
}

//__device__ vec3 ray_color(const ray& r) {
//	vec3 unit_direction = unit_vector(r.direction());
//	float t = 0.5f * (unit_direction.y() + 1.0f);
//	vec3 c = (1.0f - t) * vec3(1.0, 1.0, 1.0) + t * vec3(0.5, 0.7, 1.0);
//	return  c;
//}

__device__ vec3 ray_color(hiprandState *state,const ray& r,int depth,const hittable** world) {
	ray cur_ray = r;
	vec3 cur_attenuation = vec3(1.0, 1.0, 1.0);
	for (int i = 0; i < depth; i++) {
		hit_record rec;
		//if(false){
		if ((*world)->hit(cur_ray, 0.001f, FLT_MAX, rec)) {
			ray scattered;
			vec3 attenuation;
			if (rec.mat->scatter(cur_ray, rec, attenuation, scattered,state)) {
				cur_ray = scattered;
				cur_attenuation *= attenuation;
			}
			else {
				return vec3(0.0, 0.0, 0.0);
			}
		}
		else {
			vec3 unit_direction = unit_vector(cur_ray.direction());
			float t = 0.5f * (unit_direction.y() + 1.0f);
			vec3 c = (1.0f - t) * vec3(1.0, 1.0, 1.0) + t * vec3(0.5, 0.7, 1.0);
			return cur_attenuation * c;
		}

	}
	return vec3(0.0, 0.0, 0.0);
}
__global__ void CalculatePerPixel(hittable** world, camera** camera, hiprandState* global_rand_state, int spp, unsigned int* g_odata, int imgh, int imgw) {

	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int bw = blockDim.x;
	int bh = blockDim.y;
	int i = blockIdx.x * bw + tx;
	int j = blockIdx.y * bh + ty;
	int index = i + j * imgh;







	//ray r(camera_center, ray_direction);
	//vec3 pc = ray_color(r);

	//printf("%f %f %f\n", pc.x(), pc.y(), pc.z());
	hiprandState local_rand_state = global_rand_state[index];
	vec3 color(0, 0, 0);

	int depth = (*camera)->max_depth;
	ray r = (*camera)->get_ray(&local_rand_state, i, j);
	vec3 pc = ray_color(&local_rand_state, r, depth, world);
	//vec3 pc = ray_color(r);
	color /= float(spp);
	//g_odata[index] = rgbToInt((float)x / 800 * 255, (float)y / 800 * 255, 0);
	global_rand_state[index] = local_rand_state;
	g_odata[i + j * imgw] = vectorgb(pc);
}
__global__ void initCamera(camera** ca) {

	*ca = new camera(16.0 / 9.0, //종횡비
		1600, //이미지 가로길이
		10,  //픽셀당 샘플수
		50,  //반사 횟수
		90,  //시야각
		vec3(0, 0, 0), //카메라 위치 
		vec3(0, 0, -1), //바라보는곳
		vec3(0, 1, 0)); //업벡터

}
__global__ void movCam(camera** ca, int direction) {
	(*ca)->moveorigin(direction);
}
__global__ void RotateCam(camera** ca, vec3 direction) {

	auto beta = direction.y() / 50;


	
	auto alpha = direction.x() * 90 / 800;
	(*ca)->lookat = vec3(cos(degrees_to_radians(beta)) * sin(degrees_to_radians(alpha)), sin(degrees_to_radians(beta)), cos(degrees_to_radians(beta)) * cos(degrees_to_radians(alpha)));
	printf("현재 바라보는 방향 %f %f %f\n", (*ca)->lookat.x(), (*ca)->lookat.y(), -(*ca)->lookat.z());
	printf("  현재 바라보는 위치 %f %f %f\n", (*ca)->lookfrom.x(), (*ca)->lookfrom.y(), -(*ca)->lookfrom.z());
	(*ca)->initialize();
	//(*ca)-origin(direction);
}
__global__ void initWorld(hittable** world, hittable** objects,int object_counts) {


	objects[0] = new sphere(vec3(0, -1000.0, 0), 1000, new lambertian(vec3(0.5, 0.5, 0.5)));
	objects[1] = new sphere(vec3(0, 0, -1), 0.5, new lambertian(vec3(0.7, 0.8, 0.0)));
	//objects[0] = ground;
	*world = new hittable_list(objects, object_counts);
	//(*world)->add(ground);
}
extern "C" void initTracing() {

	hipMalloc(&cam, sizeof(camera*));
	initCamera << <1, 1 >> > (cam);
	hipMalloc((void**) &objects, object_counts * sizeof(hittable*));//오브젝트 개수만큼 할당 필요
	hipMalloc((void**)&world, sizeof(hittable*));
	initWorld << <1, 1 >> > (world, objects,object_counts);
}
extern "C" void moveCamera(int direction) {
	movCam << <1, 1 >> > (cam, direction);
}
extern "C" void RotateCamera(int x,int y) {
	RotateCam << <1, 1 >> > (cam,vec3(x,y,0));
}
__global__ void Random_Init(hiprandState* global_state, int ih) {
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int bw = blockDim.x;
	int bh = blockDim.y;
	int x = blockIdx.x * bw + tx;
	int y = blockIdx.y * bh + ty;
	unsigned int pixel_index = x + y * ih;
	hiprandState s;
	hiprand_init(pixel_index, 0, 0, &global_state[pixel_index]);
}
extern "C" void initCuda(dim3 grid, dim3 block, int image_height, int image_width, int pixels) {
	hipMalloc(&random_state, pixels * sizeof(hiprandState));
	Random_Init << <grid, block, 0 >> > (random_state, image_height);
}


extern "C" void generatePixel(dim3 grid, dim3 block, int sbytes,
	unsigned int* g_odata, int imgh, int imgw) {
	CalculatePerPixel << <grid, block, sbytes >> > (world, cam, random_state, 10, g_odata, imgh, imgw);
}