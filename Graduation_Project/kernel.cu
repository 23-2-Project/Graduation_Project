#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime_api.h"
#include <hiprand/hiprand_kernel.h>
#include <math_functions.h>
#include <hittable_list.h>
#include <ray.h>
#include <sphere.h>
#include <hittable_list.h>
#include <material.h>
#include <vec3.h>
#include <camera.h>
#include <bvh.h>
#include <triangle.h>
#include <obj.h>
#include <assimp/Importer.hpp>
#include <assimp/scene.h>
#include <assimp/postprocess.h>

hittable_list** world;
//hittable** objects;
camera** cam;
int object_counts = 29000;

hiprandState* random_state;
// convert floating point rgb color to 8-bit integer
__device__ float clamp(double x, double a, double b) { return max(a, min(b, x)); }
__device__ int rgbToInt(double r, double g, double b) {
	r = clamp(r, 0.0f, 255.0f);
	g = clamp(g, 0.0f, 255.0f);
	b = clamp(b, 0.0f, 255.0f);
	return (int(b) << 16) | (int(g) << 8) | int(r);
}
__device__ int vectorgb(vec3 color) {
	return rgbToInt(color.x()*255,color.y()*255,color.z()*255);
}
__global__ void movCam(camera** ca, int direction, int weight) {
	(*ca)->moveorigin(direction, weight);
}
__global__ void RotateCam(camera** ca, vec3 direction) {
	(*ca)->rotate(direction);
}
__global__ void ManipulateVFOV(camera** ca, int x) {
	(*ca)->changevfov(x);
}
extern "C" void moveCamera(int direction, int weight) {
	movCam << <1, 1 >> > (cam, direction, weight);
}
extern "C" void RotateCamera(int x, int y) {
	RotateCam << <1, 1 >> > (cam, vec3(x, y, 0));
}
extern "C" void manivfov(int x) {
	ManipulateVFOV << <1, 1 >> > (cam, x);
}

__global__ void CalculatePerPixel(hittable_list** world, camera** camera, hiprandState* global_rand_state, unsigned int* g_odata, int imgh, int imgw) {
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int bw = blockDim.x;
	int bh = blockDim.y;
	int i = blockIdx.x * bw + tx;
	int j = blockIdx.y * bh + ty;
	int index = i + j * imgh;

	hiprandState local_rand_state = global_rand_state[index];
	vec3 color(0, 0, 0);

	int depth = (*camera)->max_depth;
	int spp = (*camera)->samples_per_pixel;
	ray r = (*camera)->get_ray(&local_rand_state, i, j);
	for (int i = 0; i < spp; i++) {
		color += (*camera)->ray_color(&local_rand_state, r, depth, world);
	}
	color /= float(spp);
	global_rand_state[index] = local_rand_state;
	g_odata[i + j * imgw] = vectorgb(color);
}
__global__ void initCamera(camera** ca) {

	*ca = new camera(16.0 / 9.0, //종횡비
		1600, //이미지 가로길이
		1,  //픽셀당 샘플수
		50,  //반사 횟수
		90,  //시야각
		vec3(-50, 0, 0), //카메라 위치 
		vec3(0, 0, -1), //바라보는곳
		vec3(0, 1, 0)); //업벡터

}
__global__ void initWorld(hittable_list** world, int object_counts) {
	(*world) = new hittable_list(object_counts);
}
#define RND (hiprand_uniform(&local_rand_state))
__global__ void addObjects(hiprandState* global_state, hittable_list** world, int object_counts) {
	hiprand_init(0, 0, 0, &global_state[0]);
	hiprandState local_rand_state = *global_state;
	(*world)->add(new sphere(vec3(0, -1000.0, 0), 1000, new lambertian(vec3(0.5, 0.5, 0.5))));
	for (int a = -2; a < 2; a++) {
		for (int b = -2; b < 2; b++) {
			float choose_mat = RND;
			vec3 center(a + RND, 0.2, b + RND);
			if (choose_mat < 0.8f) {
				(*world)->add(new sphere(center, 0.2, new lambertian(vec3(RND * RND, RND * RND, RND * RND))));
			}
			else if (choose_mat < 0.95f) {
				(*world)->add(new sphere(center, 0.2, new metal(vec3(0.5f * (1.0f + RND), 0.5f * (1.0f + RND), 0.5f * (1.0f + RND)), 0.0f/*0.5f * RND*/)));
			}
			else {
				(*world)->add(new sphere(center, 0.2, new dielectric(1.5)));
			}
		}
	}
}
__global__ void makeBVH(hiprandState* global_state, hittable_list** world, int object_counts) {
	printf("%d개\n", (*world)->now_size);
	hiprand_init(0, 0, 0, &global_state[0]);
	hiprandState local_rand_state = *global_state;
	(*world) = new hittable_list((hittable*)new bvh_node(world, &local_rand_state), object_counts);
}

__global__ void Random_Init(hiprandState* global_state, int ih) {
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int bw = blockDim.x;
	int bh = blockDim.y;
	int x = blockIdx.x * bw + tx;
	int y = blockIdx.y * bh + ty;
	unsigned int pixel_index = x + y * ih;
	hiprandState s;
	hiprand_init(pixel_index, 0, 0, &global_state[pixel_index]);
}
extern "C" void initCuda(dim3 grid, dim3 block, int image_height, int image_width, int pixels) {
	hipMalloc(&random_state, pixels * sizeof(hiprandState));
	Random_Init << <grid, block, 0 >> > (random_state, image_height);

	//랜덤 초기화

	hipMalloc((void**)&world, sizeof(hittable*));
	initWorld << <1, 1 >> > (world, object_counts); hipDeviceSynchronize();
	//월드 초기화 OBJ 읽기 및 카메라 등
	Assimp::Importer importer;
	const aiScene* scene = importer.ReadFile("FileName.obj", aiProcess_Triangulate | aiProcess_GenSmoothNormals | aiProcess_FlipUVs | aiProcess_CalcTangentSpace);
	
	if (!scene || scene->mFlags & AI_SCENE_FLAGS_INCOMPLETE || !scene->mRootNode) // if is Not Zero
	{
		printf("Read File Exception\n");
	}
	//int cnt = 0;
	//for (int i = 0; i < scene->mNumMeshes; i++) {
	//	auto mesh = scene->mMeshes[i];
	//	for (int j = 0; j < mesh->mNumFaces; j++) {
	//		auto Face = mesh->mFaces[j];
	//		/*newTriangle(mesh->mVertices[Face.mIndices[0]].x,
	//			mesh->mVertices[Face.mIndices[0]].y,
	//			mesh->mVertices[Face.mIndices[0]].z,

	//			mesh->mVertices[Face.mIndices[1]].x,
	//			mesh->mVertices[Face.mIndices[1]].y,
	//			mesh->mVertices[Face.mIndices[1]].z,

	//			mesh->mVertices[Face.mIndices[2]].x,
	//			mesh->mVertices[Face.mIndices[2]].y,
	//			mesh->mVertices[Face.mIndices[2]].z);*/
	//	}
	//}
	





	//여기까지 OBJ 읽기
	hiprandState* objectinit;
	hipMalloc(&objectinit, sizeof(hiprandState));
	addObjects<< <1, 1 >> > (objectinit, world, object_counts);
	hipMalloc(&cam, sizeof(camera*));
	initCamera << <1, 1 >> > (cam);

	hipDeviceSynchronize();//쿠다커널이 종료될때까지 기다리는 함수. 위의 world에 오브젝트 다 담길때까지 기다리는 거임. 
	//그래야 BVH할수있으니까. BVH하다가 오브젝트 담기면 안됌.

	hiprandState* bvh_state;
	hipMalloc(&bvh_state, sizeof(hiprandState));
	makeBVH << <1, 1 >> > (bvh_state, world, object_counts);
}
extern "C" void generatePixel(dim3 grid, dim3 block, int sbytes,
	unsigned int* g_odata, int imgh, int imgw) {

	CalculatePerPixel << <grid, block, sbytes >> > (world, cam, random_state, g_odata, imgh, imgw);
}