#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime_api.h"
#include <hiprand/hiprand_kernel.h>
#include <math_functions.h>
#include <hittable_list.h>
#include <ray.h>
#include <sphere.h>
#include <hittable_list.h>
#include <material.h>
#include <vec3.h>
#include <camera.h>
#include <bvh.h>
#include <triangle.h>
#include <obj.h>
#include <assimp/Importer.hpp>
#include <assimp/scene.h>
#include <assimp/postprocess.h>

hittable_list** world;
bvh_node** bvh_list;
camera** cam;
int object_counts = 80000;
hiprandState* random_state;

// convert floating point rgb color to 8-bit integer
__device__ float clamp(double x, double a, double b) { return max(a, min(b, x)); }
__device__ int rgbToInt(double r, double g, double b) {
	r = clamp(r, 0.0f, 255.0f);
	g = clamp(g, 0.0f, 255.0f);
	b = clamp(b, 0.0f, 255.0f);
	return (int(b) << 16) | (int(g) << 8) | int(r);
}
__device__ int vectorgb(vec3 color) {
	return rgbToInt(color.x()*255,color.y()*255,color.z()*255);
}
__global__ void movCam(camera** ca, int direction, int weight) {
	(*ca)->moveorigin(direction, weight);
}
__global__ void RotateCam(camera** ca, vec3 direction) {
	(*ca)->rotate(direction);
}
__global__ void ManipulateVFOV(camera** ca, int x) {
	(*ca)->changevfov(x);
}
extern "C" void moveCamera(int direction, int weight) {
	movCam << <1, 1 >> > (cam, direction, weight);
}
extern "C" void RotateCamera(int x, int y) {
	RotateCam << <1, 1 >> > (cam, vec3(x, y, 0));
}
extern "C" void manivfov(int x) {
	ManipulateVFOV << <1, 1 >> > (cam, x);
}
extern "C" void constructBVH() {
	// object 개수 구하기
	int object_count;
	int* kernel_value;
	hipMalloc((void**)&kernel_value, sizeof(int));
	get_object_count << <1, 1 >> > (world, kernel_value);
	hipMemcpy(&object_count, kernel_value, sizeof(int), hipMemcpyDeviceToHost);
	hipFree(kernel_value);
	printf("%d개\n", object_count);
	
	// 정렬
	srand(time(NULL));
	int axis = rand() % 3;
	dim3 sortBlock(512, 1, 1);
	dim3 sortGrid(object_count / sortBlock.x + 1, 1, 1);
	for (int i = 0; i < object_count; ++i) {
		int odd_even = i % 2;
		object_swap << <sortGrid, sortBlock>> > (world, object_count, odd_even, 0);
		hipDeviceSynchronize();
	}
	printf("정렬 완료\n");

	// 할당
	int startIdx = 1 << 30;
	while (true) {
		if ((startIdx >> 1) > object_count) { startIdx >>= 1; }
		else { break; }
	}

	hipMalloc((void**)&bvh_list, (startIdx * 2) * sizeof(bvh_node*));
	dim3 bvhBlock(512, 1, 1);
	dim3 bvhGrid(startIdx * 2 / bvhBlock.x + 1, 1, 1);
	add_bvh_node << <bvhGrid, bvhBlock>> > (bvh_list, startIdx * 2);
	printf("할당 완료\n");


	//bvh 생성
	hiprandState* bvh_state;
	hipMalloc(&bvh_state, sizeof(hiprandState));
	make_bvh_tree << <1, 1 >> > (bvh_state, world, bvh_list, object_count);
	printf("bvh 생성 완료\n");
}

__global__ void CalculatePerPixel(hittable_list** world, camera** camera, hiprandState* global_rand_state, unsigned int* g_odata, int imgh, int imgw) {
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int bw = blockDim.x;
	int bh = blockDim.y;
	int i = blockIdx.x * bw + tx;
	int j = blockIdx.y * bh + ty;
	int index = i + j * imgh;

	hiprandState local_rand_state = global_rand_state[index];
	vec3 color(0, 0, 0);

	int depth = (*camera)->max_depth;
	int spp = (*camera)->samples_per_pixel;
	ray r = (*camera)->get_ray(&local_rand_state, i, j);
	for (int i = 0; i < spp; i++) {
		color += (*camera)->ray_color(&local_rand_state, r, depth, world);
	}
	color /= float(spp);
	global_rand_state[index] = local_rand_state;
	g_odata[i + j * imgw] = vectorgb(color);
}
__global__ void initCamera(camera** ca) {
	*ca = new camera(16.0 / 9.0, //종횡비
		1600, //이미지 가로길이
		1,  //픽셀당 샘플수
		50,  //반사 횟수
		90,  //시야각
		vec3(-20, 0, 0), //카메라 위치 
		vec3(0, 0, -1), //바라보는곳
		vec3(0, 1, 0)); //업벡터
}
__global__ void initWorld(hittable_list** world, int object_counts) {
	(*world) = new hittable_list(object_counts);
}
#define RND (hiprand_uniform(&local_rand_state))
__global__ void addObjects(hiprandState* global_state, hittable_list** world, int object_counts) {
	hiprand_init(0, 0, 0, &global_state[0]);
	hiprandState local_rand_state = *global_state;
	(*world)->add(new sphere(vec3(0, -1000.0, 0), 1000, new lambertian(vec3(0.5, 0.5, 0.5))));

	for (int a = 0; a < 0; a++) {
		for (int b = -10; b < 10; b++) {
			float choose_mat = RND;
			vec3 center(a + RND, 0.2, b + RND);
			if (choose_mat < 0.8f) {
				(*world)->add(new sphere(center, 0.2, new lambertian(vec3(RND * RND, RND * RND, RND * RND))));
			}
			else if (choose_mat < 0.95f) {
				(*world)->add(new sphere(center, 0.2, new metal(vec3(0.5f * (1.0f + RND), 0.5f * (1.0f + RND), 0.5f * (1.0f + RND)), 0.0f/*0.5f * RND*/)));
			}
			else {
				(*world)->add(new sphere(center, 0.2, new dielectric(1.5)));
			}
		}
	}
}

__global__ void addTriangle(hittable_list** world,vec3 a,vec3 b,vec3 c) {
	(*world)->add(new triangle(a, b, c, new lambertian(vec3(0.5f, 0.0f, 0.0f))));
}
__global__ void Random_Init(hiprandState* global_state, int ih) {
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int bw = blockDim.x;
	int bh = blockDim.y;
	int x = blockIdx.x * bw + tx;
	int y = blockIdx.y * bh + ty;
	unsigned int pixel_index = x + y * ih;
	hiprand_init(pixel_index, 0, 0, &global_state[pixel_index]);
}
extern "C" void initCuda(dim3 grid, dim3 block, int image_height, int image_width, int pixels) {
	hipMalloc(&random_state, pixels * sizeof(hiprandState));
	Random_Init << <grid, block, 0 >> > (random_state, image_height);

	//랜덤 초기화
	hipMalloc((void**)&world, sizeof(hittable*));
	initWorld << <1, 1 >> > (world, object_counts); hipDeviceSynchronize();

	//월드 초기화 OBJ 읽기 및 카메라 등
	Assimp::Importer importer;
	const aiScene* scene = importer.ReadFile("chair.obj", aiProcess_Triangulate | aiProcess_GenSmoothNormals | aiProcess_FlipUVs | aiProcess_CalcTangentSpace);
	
	if (!scene || scene->mFlags & AI_SCENE_FLAGS_INCOMPLETE || !scene->mRootNode) // if is Not Zero
	{
		printf("Read File Exception\n");
	}
	for (int i = 0; i < scene->mNumMeshes; i++) {
		auto mesh = scene->mMeshes[i];
		for (int j = 0; j < mesh->mNumFaces; j++) {
			auto Face = mesh->mFaces[j];
			addTriangle << <1, 1 >> > (world,
				vec3(mesh->mVertices[Face.mIndices[0]].x, mesh->mVertices[Face.mIndices[0]].y, mesh->mVertices[Face.mIndices[0]].z),
				vec3(mesh->mVertices[Face.mIndices[1]].x, mesh->mVertices[Face.mIndices[1]].y, mesh->mVertices[Face.mIndices[1]].z),
				vec3(mesh->mVertices[Face.mIndices[2]].x, mesh->mVertices[Face.mIndices[2]].y, mesh->mVertices[Face.mIndices[2]].z));
			/*newTriangle(mesh->mVertices[Face.mIndices[0]].x,
				mesh->mVertices[Face.mIndices[0]].y,
				mesh->mVertices[Face.mIndices[0]].z,

				mesh->mVertices[Face.mIndices[1]].x,
				mesh->mVertices[Face.mIndices[1]].y,
				mesh->mVertices[Face.mIndices[1]].z,

				mesh->mVertices[Face.mIndices[2]].x,
				mesh->mVertices[Face.mIndices[2]].y,
				mesh->mVertices[Face.mIndices[2]].z);*/
		}
	}
	
	//여기까지 OBJ 읽기
	hiprandState* objectinit;
	hipMalloc(&objectinit, sizeof(hiprandState));
	addObjects<< <1, 1 >> > (objectinit, world, object_counts);
	hipMalloc(&cam, sizeof(camera*));
	initCamera << <1, 1 >> > (cam);

	hipDeviceSynchronize();//쿠다커널이 종료될때까지 기다리는 함수. 위의 world에 오브젝트 다 담길때까지 기다리는 거임. 
	//그래야 BVH할수있으니까. BVH하다가 오브젝트 담기면 안됌.

	constructBVH();
}
extern "C" void generatePixel(dim3 grid, dim3 block, int sbytes,
	unsigned int* g_odata, int imgh, int imgw) {
	CalculatePerPixel << <grid, block, sbytes >> > (world, cam, random_state, g_odata, imgh, imgw);
}